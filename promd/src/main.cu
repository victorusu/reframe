#include "hip/hip_runtime.h"
#include <iostream>  // For output to terminal
#include <fstream>   // For file I/O
#include <sstream>   // For file I/O
#include <iomanip>   // For output format
#include <cmath>     // For atan(), sqrt() etc.
#include <ctime>     // For timing functions
#include <string>    // For string manipulation
#include <exception> // For array manipulation

#include "main.hpp"  // For function prototypes and run-time constants.

// #include "configuration/conf.hpp"
#include "domain/domain.hpp"
#include "utilities/getopt.hpp"

#include "interaction/pairlist/pairlist.hpp"
#include "interaction/nonbonded/nonbonded.hpp"

#include "simulation/energies.hpp"

#include "simulation/integrator/integrator.hpp"
#include "simulation/thermostat/thermostat.hpp"
#include "simulation/barostat/barostat.hpp"
#include "simulation/gather.hpp"

#include "interaction/bonded/bonded.hpp"

#include "topology/molecules.hpp"

#include "files/imdfilereader.hpp"
#include "files/topologyfilereader.hpp"
#include "files/trajfiles.hpp"
#include "files/omdfile.hpp"

int main(int argc, char *argv[])
{
    startTimer(totalProgramTimer);

    dd.init(argc, argv);
    // cerr.setDomainDecomposition(dd);
    // cout.setDomainDecomposition(dd);

    GetOpt getOpt(argc, argv);

    getOpt.help = "Help intruction";
    getOpt  << ArgumentOptions("-nt",  "number of processors",                          ArgumentOptions::INT,     "max available", "help", false, true, false)
            << ArgumentOptions("-imd", "simulation parameters file",                    ArgumentOptions::STRING,     "system.imd", "help", false, true, false)
            << ArgumentOptions("-cnf", "coordinate file. Formats: gro, cnf",            ArgumentOptions::STRING,     "system.cnf", "help", false, true, false)
            << ArgumentOptions("-top", "topology file",                                 ArgumentOptions::STRING,     "system.top", "help", false, true, false)
            << ArgumentOptions("-out", "output coordinate file Formats: gro, cnf, g96", ArgumentOptions::STRING,     "system.g96", "help", false, true, false)
            << ArgumentOptions("-trc", "trajectory file Formats: gro, cnf, g96, trc",   ArgumentOptions::STRING,     "system.trc", "help", false, true, false)
            << ArgumentOptions("-trf", "trajectory file Formats: trf",                  ArgumentOptions::STRING,     "system.trf", "help", false, true, false)
            << ArgumentOptions("-omd", "log file",                                      ArgumentOptions::STRING,     "system.omd", "help", false, true, false);

    // Checking for help before anything else
    std::string help = getOpt.parse("-h");
    if(help != "false") {
        getOpt.printHelp();
        dd.finalize();
        return 0;
    }

    // Setting the number of processors per mpi rank
    // each mpi rank should take the maximum available per node
    // if the nprocs was not defined by the user
    {
        int nprocs = 1;

        std::string opt = getOpt.parse("-nt");
        if(!opt.empty())
            nprocs = atoi(opt.c_str());

#if defined(_OPENMP)
        if(nprocs < 1) {
            dd.setNumberProcs();
            nprocs = dd.nprocs;
        }
        else {
            dd.setNumberProcs(nprocs);
        }
        omp_set_num_threads(dd.nprocs);
#else
        if(nprocs != 0)
            cerr.printf("Cannot set the number of threads to %d for you are not running an OpenMP binary\n", nprocs);
        dd.nprocs = 1;
#endif
    }
    // blaze::setNumThreads(dd.nprocs);

    cerr.printf("Number of processors: %d\n"
                "          Running on: %s\n\n", dd.nprocs, dd.hostname);

    const std::string trcfilename = getOpt.parse("-trc");
    const std::string trffilename = getOpt.parse("-trf");
    const std::string omdfilename = getOpt.parse("-omd");
    const std::string cnffilename = getOpt.parse("-cnf");
    const std::string imdfilename = getOpt.parse("-imd");
    const std::string topfilename = getOpt.parse("-top");

    IMDFileReader       inputimdFile;             // simulation parameters
    OMDFile             omdfile;                  // omd file
    TOPOLOGYFileReader  topfile;                  // topo file
    TRFFileWriter       trffile;

    InputTraj  *inputCONFFile = NULL;    // input coordinate
    OutputTraj *outputTRAJFile = NULL;   // trajectory file


    SimParameters simParam;
    Configuration conf;
    Box simBox;
    PairList pairlist;
    // pairlist.nprocs = dd.nprocs;

    Nonbonded nonbonded;
    Bonded bonded;
    Integrator integrator;
    Topology topology;

    Thermostat *thermostat = NULL;
    Barostat *barostat = NULL;
    SHAKE shake;

    Energies energies;

    std::stringstream ss;

    G96FileWriter cellDebugger;

    //
    // Opening the OMD file
    //
    // This should allow us to print the evolution of the MD
    //
    if(dd.master()) {
        if(!omdfile.open(omdfilename)) {
            cerr.printf("Unable to open file: %s\n", omdfile.getFileName().c_str());
            dd.abort();
            return 1;
        }
    }

    omdfile.printf("Running %s with %d thread(s)\n\n", argv[0], dd.nprocs);
    omdfile.printf("Precision: %s\n", prec);
    omdfile.printf("Time Unit: %s\n\n", timeUnit);

    cout.printf("Running %s with %d thread%s\n\n", argv[0], dd.nprocs, dd.nprocs > 1 ? "(s)" : "");
    cout.printf("Precision: %s\n", prec);
    cout.printf("Time Unit: %s\n\n", timeUnit);

    omdfile.printf("\n PROGRAM SENSITIVITY ANALISES PERFORMS A MD-RUN\n\n");

    //
    // Reading the IMD file
    //
    // This should read the simulation conditions
    //
    if(!inputimdFile.open(imdfilename)) {
        cerr.printf("Unable to open file: %s\n", inputimdFile.getFileName().c_str());
        dd.abort();
        return 1;
    }
    if(!inputimdFile.readfile(simParam)) {
        cerr.printf("Unable to read imd file: %s\n", inputimdFile.getFileName().c_str());
        dd.abort();
        return 1;
    }
    inputimdFile.close();

    //
    // Printing the imd file
    //
    //
    omdfile.printf("=============================================\n");
    omdfile.printf(" R E A D I N G   T H E   I N P U T   D A T A\n");
    omdfile.printf("=============================================\n\n");

    omdfile.printSimParameters(simParam);

    //
    // Reading the topology file
    //
    // This should read the topology and the physical constants
    // The latter should be save at simParam
    //
    if(!topfile.open(topfilename)) {
        cerr.printf("Unable to open file: %s\n", topfilename.c_str());
        dd.abort();
        return 1;
    }
    if(!topfile.readfile(conf, simBox, topology, simParam)) {
        cerr.printf("Unable to read top file: %s\n", topfilename.c_str());
        dd.abort();
        return 1;
    }
    topfile.close();


    // cerr.printf("topology.bondAngleTypes: %d\n", topology.bondAngleTypes.ntypes);
    // int i;
    // for(i = 0; i < topology.bondAngleTypes.ntypes; i++ ) {
    //     cerr.printf("topology.bondAngleTypes[%d] %f %f %f\n", i, topology.bondAngleTypes.kq[i], topology.bondAngleTypes.kh[i], topology.bondAngleTypes.a0[i]);
    // }

    // cerr.printf("topology.bondAngles: %d\n", topology.bondAngles.nBondAngles);
    // for(i = 0; i < topology.bondAngles.nBondAngles; i++ ) {
    //     cerr.printf("topology.bondAngle[%d] has atoms %d %d %d type %d force constant %f and cos0 %f\n", i, topology.bondAngles.atoms[3*i], topology.bondAngles.atoms[3*i+1], topology.bondAngles.atoms[3*i+2], topology.bondAngles.type[i], topology.bondAngles.kq[i], topology.bondAngles.cos0[i]);
    // }

    // // return 0;


    //
    // Allocating the thermostat
    //
    // This must be done after the topology reading
    //
    if(!allocateThermostat(&thermostat, simParam, topology.nAtoms)) {
        cerr.printf("Could not identify thermostat\n");
        dd.abort();
        return 1;
    }

    //
    // Allocating the barostat
    //
    // We just read if we have thermostat or not
    //
    if(!allocateBarostat(&barostat, simParam)) {
        cerr.printf("Could not identify barostat\n");
        dd.abort();
        return 1;
    }

    //
    // Reading the coordinates and velocities file
    //
    // We should check the size of the vectors. Specially the force vectors
    //
    if(!allocateInputTRAJFile(&inputCONFFile, cnffilename))
    {
        cerr.printf("Unable to allocate file %s of type: %s\n", cnffilename.c_str(), FileHandler::getFileNameExt(cnffilename).c_str());
        dd.abort();
        return 1;
    }
    if(!inputCONFFile->open()) {
        cerr.printf("Unable to open file: %s\n", cnffilename.c_str());
        dd.abort();
        return 1;
    }

    // Reserving memory for the configuration before reading
    conf.nAtoms = topology.nAtoms;
    // conf.init();
    conf.reserve(topology.nAtoms, dd.nprocs);

    // This reading should already place the atoms inside the box
    if(!inputCONFFile->readfile(conf, simBox, topology, simParam)) {
        cerr.printf("Number of atoms is different in TOP and CNF files\n");
        dd.abort();
        return 1;
    }
    inputCONFFile->close();

    //
    // Correcting the force vector size
    //
    // In this implementation we allocate it to be: dd.nprocs * nAtoms
    //
    // conf.correctForceAllocation(dd.nprocs);
    conf.correctAllocation(dd.nprocs);


    // updating the constrained atoms list
    if(simParam.shake)
        topology.populateConstrainedAtoms();

    //
    // Finally we check the simulation parameters combined with the simulation box
    //
    //
    if(!checkSimParam(simParam, simBox)) {
        dd.abort();
        return 1;
    }
    // omdfile.printf("Done!\n");



    //
    // All the reads are done so we open the trajectory file
    //
    if(dd.master()) {
        if(!allocateOutputTRAJFile(&outputTRAJFile, trcfilename))
        {
            cerr.printf("Unable to allocate output file type: %s\n", FileHandler::getFileNameExt(trcfilename).c_str());
            dd.abort();
            return 1;
        }
        if(simParam.ntwx) {
            if(!outputTRAJFile->open()) {
                cerr.printf("Unable to open file: %s\n", trcfilename.c_str());
                dd.abort();
                return 1;
            }
            else {
                outputTRAJFile->writeTitle(conf.title);
            }
        }
        // if(!outputTRAJFile->writeFile(conf, simBox)) {
        //     cerr.printf("Unable to create file: %s\n", trcfilename.c_str());
        //     dd.abort();
        //     return 1;
        // }
    }

    if(dd.master()) {
        if(simParam.ntwf) {
            trffile.setFileName(trffilename);
            if(!trffile.open()) {
                cerr.printf("Unable to open file: %s\n", trffilename.c_str());
                dd.abort();
                return 1;
            } else {
                trffile.writeTitle(conf.title);
            }
        }

    }


    // preparing the SHAKE
    if(simParam.shake) {
        shake.prepare(topology);
    }

    // preparing the energies
    energies.extend(simParam.nstlim);


    //
    // We should generate Maxwell-Boltzmann distribution of velocities if requested
    //
    // But I haven't implemented it yet
    //
    omdfile.printf("\n===================================================================\n");
    omdfile.printf(" A T O M I C   C O O R D I N A T E S   A N D   V E L O C I T I E S\n");
    omdfile.printf("===================================================================\n\n");

    omdfile.printf("\nShould print here the coordinates and possibly generated velocities!\n");
    omdfile.printf("But I haven't implemented it yet\n\n");
    omdfile.printf("Done!\n");




    omdfile.printf("\n===================================\n");
    omdfile.printf(" P A I R L I S T   C R E A T I O N\n");
    omdfile.printf("===================================\n\n");

    //
    // Splitting box into "domains"
    //
    // Well, it was implemented like domain, now it is just a grid! :(
    //
    //
    const bool gridOK = dd.splitBoxIntoCells(conf.nAtoms, simParam.rlist, simBox);

    if(!gridOK && dd.master()) {
        cerr.printf("\nUnable to split the box into cells.\n"
            "Please decrease the number of processors or change the box size.\n\n");

        cerr.printf("current Cell decomposition: %dx%dx%d\n", dd.nCells[0], dd.nCells[1], dd.nCells[2]);
        omdfile.printf("current Cell decomposition: %dx%dx%d\n", dd.nCells[0], dd.nCells[1], dd.nCells[2]);

        omdfile.close();

        dd.abort();
        return 1;
    }

    // Printing the grid info
    omdfile.printf("     shifts: %dx%dx%d\n", dd.shifts[0], dd.shifts[1], dd.shifts[2]);
    omdfile.printf("nCellsTotal: %dx%dx%d\n", dd.nTotalCellsX(), dd.nTotalCellsY(), dd.nTotalCellsZ());
    omdfile.printf("nTotalCells: %d\n", dd.nTotalCells());

    cerr.printf("     shifts: %dx%dx%d\n", dd.shifts[0], dd.shifts[1], dd.shifts[2]);
    cerr.printf("nCellsTotal: %dx%dx%d\n", dd.nTotalCellsX(), dd.nTotalCellsY(), dd.nTotalCellsZ());
    cerr.printf("nTotalCells: %d\n", dd.nTotalCells());


    // Statistics on the pairlist update
    int stepsWithoutPairListUpdate = 0;
    int totalStepsWithoutPairListUpdate = 0;
    int numberPairListUpdate = 0;

    // Checking whether we are updating based on a fix frequency time or based on molecular motion
    const double halfRlistDiff = (simParam.rlist - std::max(simParam.rcutvdw, simParam.rcutcoul)) * 0.5;
    if(simParam.nstlist < 0) {
        pairlist.doMaxDisplacement = true;
        pairlist.maxDisplacement = 0.0;
        if(halfRlistDiff < 0.0) {
            cerr.printf("RLIST must be greater than RCUTF in order to set NSTLIST < 1\n");
            return 1;
        }
        omdfile.printf("Updating pairlist if displacements are greater than %f\n", halfRlistDiff);
        cerr.printf("Updating pairlist if displacements are greater than %f\n", halfRlistDiff);
    }
    else {
        pairlist.doMaxDisplacement = false;
        omdfile.printf("\nUpdating pairlist every: %d steps\n", simParam.nstlist);
        cerr.printf("\nUpdating pairlist every: %d steps\n", simParam.nstlist);
    }

    // Creating the pairlist
    pairlist.create(conf, simBox, topology, true);
    // pairlist.print(omdfile, 0);

    if(pairlist.doMaxDisplacement) {
        pairlist.update(conf, simBox, topology);
        numberPairListUpdate++;
    }


    // {
    //     // int i = ii;
    //     // int j = jj;
    //     // if(i > j) {
    //     //     std::swap(i,j);
    //     // }
    //     int i, j;
    //     for(i = 0; i < topology.excl.size(); i++) {
    //         for(j = i+1; j < topology.excl.size(); j++) {
    //             if((j-i < 32) && isexcluded(topology.excl, i, j-i-1)) {
    //             // if((j-i > 31) || notexcluded(topology.excl, i, j-i-1)) {
    //                 omdfile.printf("atom %d is excluded from %d\n", i, j);
    //             }
    //         }
    //     }

    // }

    // return 0;

    cerr.printf("\nPairList creation time: %4.3f %s\n\n", pairListCreateTime, timeUnit);
    omdfile.printf("\nPairList creation time: %4.3f %s\n\n", pairListCreateTime, timeUnit);

    omdfile.printf("\n===========================\n");
    omdfile.printf(" M D   S I M U L A T I O N\n");
    omdfile.printf("===========================\n\n");


    cerr.printf("Simulation time: %f ps\n\n", simParam.nstlim * simParam.dt);
    omdfile.printf("Simulation time: %f ps\n\n", simParam.nstlim * simParam.dt);

    // calc number of dof

    // compute total mass of the system
    //CALL CLCMAS(NPM,NSM,NSPM,NSP,TOTMAS,TMASS,SUBMAS,SUBMIN)

    // calc the bath
    // CALL CLCBTH(NDOF,NBATH,NBNUM,NBNDX,EKREF,DTBATH,TFACBT)

    // int i, j;
    // for(i = 0; i < topology.excl.size(); i++) {
    //     for(j = 0; j < 32; j++)
    //         if(isexcluded(topology.excl, i, j))
    //             cerr.printf("atom %d is excluded from: %d\n", i, i+j+1);
    // }


    // int i = 0;
    // int lower = topology.atomTypes[i];
    // int max = topology.atomTypes[i];

    // // topology.atomTypes[ii];

    // int iac = ((max+1)*max)/2 + lower;
    // LJParameters lj = topology.ljParameters[iac];
    // number c12 = lj.c12;
    // number c6 = lj.c6;

    // cerr.printf("atom %d, type: %d, lj pos: %d with c12: %13.6e and c6: %13.6e\n", i, lower, iac, c12, c6);
    // return 0;

    // cerr.printf("computeKineticEnergyAndTemperature\n");
    // cerr.flush();
    integrator.computeKineticEnergyAndTemperature(conf, topology, simParam, thermostat->invBoltz);
    // cerr.printf("computeKineticEnergyAndTemperature\n");
    // cerr.flush();




    // Remove initial COM motion
    // integrator.removeCOMMotion(conf, topology, simBox);

    // **************************************************
    // main MD loop
    startTimer(mdLoopTimer);
    for(integrator.step = 0; integrator.step < simParam.nstlim; integrator.step++) {


        // if(integrator.step > 0)
        //     return 0;

        // cerr.printf("integrator.step: %d\n", integrator.step);
        // cerr.flush();

        // TODO
        // review this comment
        // Place particles back into box if necessary!
        // We can do it here, or we can do it after propagating the positions
        // We chose to do it when we read the conf file, before we create the pairlist (requirement)
        // and to replace atoms inside the box after propagating the positions

        // cerr.printf("gather\n");
        // cerr.flush();
        gather(conf, simBox, topology);
        // cerr.printf("gather\n");
        // cerr.flush();

        // cerr.printf("begin of gatherAndComputePerMoleculeCOMAndCOMV\n");
        // cerr.flush();
        // cerr.printf("gatherAndComputePerMoleculeCOMAndCOMV\n");
        // cerr.flush();
        // gatherAndComputePerMoleculeCOMAndCOMV(conf, simBox, topology, simParam, thermostat->invBoltz);
        // cerr.printf("gatherAndComputePerMoleculeCOMAndCOMV\n");
        // cerr.flush();

        // cerr.printf("end of gatherAndComputePerMoleculeCOMAndCOMV\n");
        // cerr.flush();

        // compute initial kinetic energy
        // integrator.computeKineticEnergyAndTemperature(conf.vx, conf.vy, conf.vz, topology.masses, simParam.boxdof, thermostat->invBoltz, conf.nAtoms);

        // cout.printf("kineticEnergy: %f and temperature: %f\n", conf.current->kineticEnergy, conf.current->temperature);



        // cerr.printf("outputTRAJFile->writeFile\n");
        // cerr.flush();

        // Prepare virial calculation if necessary
        // IF (LDOVIR) THEN
        //     CALL PRPVIR(NATTOT,NPM,NSM,X,V,XR,TMASS,NSPM,NSP,SUBMAS,EKCM,EKCMTO,LEVERY)
        // ENDIF

        // update pair list
        // cerr.printf("pairlist\n");
        // cerr.flush();
        // outputTRAJFile->writeFile(conf, topology, simBox, integrator.step * simParam.dt, integrator.step);
        // outputTRAJFile->flush();

        // cellDebugger.writeCells(conf, topology, simBox, pairlist, integrator.step * simParam.dt, integrator.step);

        if(pairlist.doMaxDisplacement) {

            if(pairlist.maxDisplacement > halfRlistDiff) {

                // Update statistics on the pairlist update
                totalStepsWithoutPairListUpdate += stepsWithoutPairListUpdate;
                stepsWithoutPairListUpdate=0;
                numberPairListUpdate++;

                pairlist.update(conf, simBox, topology);
            }
            else
                stepsWithoutPairListUpdate++;

        } else if (integrator.step % simParam.nstlist == 0) {

            // Update statistics on the pairlist update
            totalStepsWithoutPairListUpdate += stepsWithoutPairListUpdate;
            stepsWithoutPairListUpdate=0;
            numberPairListUpdate++;

            pairlist.update(conf, simBox, topology);

        }
        else
        {
            stepsWithoutPairListUpdate++;
        }

        // cerr.printf("pairlist\n");
        // cerr.flush();

        // COMPUTE FORCES
        // computed bonded and nonbonded interactions

        // cerr.printf("nonbonded\n");
        // cerr.flush();
        nonbonded.compute(conf, topology, pairlist, simParam, simBox, barostat->compute);
        // cerr.printf("nonbonded\n");
        // cerr.flush();

        // cerr.printf("bonded\n");
        // cerr.flush();
        bonded.compute(conf, topology, simParam, simBox, barostat->compute);
        // cerr.printf("bonded\n");
        // cerr.flush();

        // write pos and box at time t to trajectory and velocities at time t - dt/2
        // cerr.printf("outputTRAJFile->writeFile\n");
        // cerr.flush();
        if (simParam.ntwx && ((integrator.step % simParam.ntwx) == 0)) {
            outputTRAJFile->writeFile(conf, topology, simBox, integrator.step * simParam.dt, integrator.step);
            outputTRAJFile->flush();
        }

        if (simParam.ntwf && ((integrator.step % simParam.ntwf) == 0)) {
            trffile.writeFile(conf, topology, simBox, integrator.step * simParam.dt, integrator.step);
            trffile.flush();
        }

        // ss << "time step: " << integrator.step;
        // conf.title = ss.str();
        // ss.str(std::string());

        // outputTRAJFile->writeFile(conf, topology, simBox);

        // outputTRAJFile->flush();
        // return 0;


    // {
    //     cerr.printf("pressute tensor\n");
    //     int i, j;
    //     for(i = 0; i < 3; i++) {
    //         for(j = 0; j < 3; j++) {
    //             cerr.printf("%f  ", barostat->pressureTensor(i, j));
    //         }
    //         cerr.printf("\n");
    //     }
    //     cerr.printf("kineticEnergy tensor\n");
    //     for(i = 0; i < 3; i++) {
    //         for(j = 0; j < 3; j++) {
    //             cerr.printf("%f  ", conf.current->kineticEnergyTensor(i, j));
    //         }
    //         cerr.printf("\n");
    //     }
    //     cerr.printf("virial tensor\n");
    //     for(i = 0; i < 3; i++) {
    //         for(j = 0; j < 3; j++) {
    //             cerr.printf("%f  ", conf.current->virialTensor(i, j));
    //         }
    //         cerr.printf("\n");
    //     }
    // }

        // Compute virial
        // IF (LDOVIR) THEN
        //    CALL CLCVIR(EKCM,VIR,PRES,EKCMTO,VIRTOT,PRESTO)
        // ENDIF
        // cerr.printf("pressureCalculation\n");
        // cerr.flush();
        barostat->pressureCalculation(conf, simBox, topology);
        // cerr.printf("pressureCalculation\n");
        // cerr.flush();


        // computing the scaling factor of Berendsen's thermostat
        // cerr.printf("thermostat->computeScale\n");
        // cerr.flush();
        thermostat->computeScale(conf.current->kineticEnergy);
        // cerr.printf("thermostat->computeScale\n");
        // cerr.flush();

        // *************************************************************************************** \\
        // START OF LEAP FROG STEP
        // propagate velocities (unconstrained) computing the center of mass velocity
        // cerr.printf("propagateVelocities\n");
        // cerr.flush();
        integrator.propagateVelocities(conf, topology, simParam);
        // cerr.printf("propagateVelocities\n");
        // cerr.flush();

        // do the following calculations
        // compute the virial, pressure, box scaling factors at the same time
        // compute the temperature scaling factor, correct the velocities at t+dt/2
        // compute the temperature and at t-dt/2, t and t+dt/2

        // scale the velocities if coupled to a bath
        // TODO integrate the computeKineticEnergyAndTemperature with the prograpate velocties
        // integrator.computeKineticEnergyAndTemperature(conf.vx, conf.vy, conf.vz, topology.masses, simParam.boxdof, thermostat->invBoltz, conf.nAtoms);
        // cerr.printf("scaleVelocities\n");
        // cerr.flush();
        thermostat->scaleVelocities(conf);
        // cerr.printf("scaleVelocities\n");
        // cerr.flush();

        // propagate coordinates (unconstrained)
        // cerr.printf("propagatePositions\n");
        // cerr.flush();
        integrator.propagatePositions(conf, pairlist, simBox, simParam.dt);
        // cerr.printf("propagatePositions\n");
        // cerr.flush();

        // END OF LEAP FROG STEP
        // *************************************************************************************** //

        // apply shake
        // cerr.printf("apply shake\n");
        // cerr.flush();
        if(simParam.shake) {
            if(!shake.apply(conf, topology, simParam, simBox)) {
                cerr.printf("Step: %d\n", integrator.step);
                cerr.flush();

                ss << "time step: " << integrator.step;
                conf.title = ss.str();
                ss.str(std::string());

                outputTRAJFile->writeFile(conf, topology, simBox, integrator.step * simParam.dt, integrator.step);
                outputTRAJFile->flush();

                omdfile.printf("SHAKE error at step: %d\n", integrator.step);
                omdfile.flush();
                dd.abort();
            }
        }
        // cerr.printf("apply shake\n");
        // cerr.flush();


        // calculate the constrained velocities

        // calculate the kinetic energies
        // C calc temperatures
        //          DO 1110 II=1,NFTMAX
        //             IF (TFACPR(II) .GE. EPS) THEN
        //                TEMP(II) = EKNOW(II)/TFACPR(II)
        //             ELSE
        //                TEMP(II) = 0.0
        //             ENDIF
        //  1110    CONTINUE

        // cerr.printf("computeKineticEnergyAndTemperature\n");
        // cerr.flush();
        integrator.computeKineticEnergyAndTemperature(conf, topology, simParam, thermostat->invBoltz);
        // cerr.printf("computeKineticEnergyAndTemperature\n");
        // cerr.flush();

        // cerr.printf("STEP %d\n", integrator.step);
        // cerr.printf("    conf.old->kineticEnergy: %f\n", conf.old->kineticEnergy);
        // cerr.printf("conf.current->kineticEnergy: %f\n", conf.current->kineticEnergy);
        // cerr.printf("      average kineticEnergy: %f\n", 0.5 * (conf.current->kineticEnergy + conf.old->kineticEnergy));

        // cerr.printf("    conf.old->temperature: %f\n", conf.old->temperature);
        // cerr.printf("conf.current->temperature: %f\n", conf.current->temperature);
        // cerr.printf("      average temperature: %f\n", 0.5 * (conf.current->temperature + conf.old->temperature));

        // omdfile.printf("STEP %d\n", integrator.step);
        // omdfile.printf("    conf.old->kineticEnergy: %f\n", conf.old->kineticEnergy);
        // omdfile.printf("conf.current->kineticEnergy: %f\n", conf.current->kineticEnergy);
        // omdfile.printf("      average kineticEnergy: %f\n", 0.5 * (conf.current->kineticEnergy + conf.old->kineticEnergy));

        // omdfile.printf("    conf.old->temperature: %f\n", conf.old->temperature);
        // omdfile.printf("conf.current->temperature: %f\n", conf.current->temperature);
        // omdfile.printf("      average temperature: %f\n", 0.5 * (conf.current->temperature + conf.old->temperature));

        //
        // integrator.putMoleculesBackIntoBox(conf, pairlist, simBox, simParam.dt);


        // Now the conf is update to the new constrained velocities
        // integrator.computeKineticEnergyAndTemperature(conf.vx, conf.vy, conf.vz, topology.masses, simParam.boxdof, thermostat->invBoltz, conf.nAtoms);
        // integrator.computeKineticEnergyAndTemperature(conf, topology, simParam, thermostat->invBoltz);
        // cerr.printf("temperature: %f, invBoltz: %f, kineticEnergy: %f\n", integrator.temperature, thermostat->invBoltz, integrator.kineticEnergy);

        // C rescale coords if we have pressure coupling
        //          IF (NTP .NE. NTPOFF) THEN
        //             CALL SCLCRD(NATTOT,X,XC,PRES,PRESTO)
        //          ENDIF
        barostat->scaleBoxAndCoordinates(conf, simBox, simParam, pairlist);

        // write energy blocks, volume, pressure and pressure scaling
        // C writing of energies and volume,pressure etc. to energy trajectory
        //          IF (NTWE .NE. 0) THEN
        //             IF (MOD(NSTEP,NTWE) .EQ. 0) THEN
        //                CALL WRTIME(IUTRJE,LFORM,NSTEP,TIME)
        //                CALL WRNRG(IUTRJE,LFORM,
        //      $              MXEWRT,ENER,
        //      $              MXCTBL,ENERES,
        //      $              NUSNRE,EPLJ,EPEL,EPRF,EPRC)
        //                CALL WRVPRT(IUTRJE,LFORM,MXVWRT,VOLPRT)
        //             ENDIF
        //          ENDIF
        // write output, if requested
        if (simParam.ntpr && ((integrator.step % simParam.ntpr) == 0)) {
            omdfile.printStep(conf, simParam, nonbonded, bonded, simBox, integrator, *barostat);
        }

        // C add energies, volprt and temperatures to averages and average square
        //          DO 210 II=1,MXETBL
        //             DTMP = ENER(II)
        //             EPSUM(II) = EPSUM(II) + DTMP
        //             EPSQ(II)  = EPSQ(II)  + DTMP**2
        //  210     CONTINUE


        // C centre of mass printing (and removal if necessary)
        //          LREMCM = (NSCM .NE. 0)
        //          IF (LREMCM) THEN
        //             LREMCM = (MOD(NSTEP+1,NSCM) .EQ. 0)
        //          ENDIF

        //          LPRLSQ = (NTPL .NE. 0)
        //          IF (LPRLSQ) THEN
        //             LPRLSQ = (MOD(NSTEP+1,NTPL) .EQ. 0)
        //          ENDIF

        //          IF (LREMCM .OR. LPRLSQ) THEN
        //             DO 80 I3 = 1,NATTO3
        //                F(I3) = X(I3) - V(I3)*DTHALF
        //  80         CONTINUE
        //          ENDIF

        //          IF (LREMCM) THEN
        //             CALL CENMAS(NATTOT,NPM,NRP,NSM,NRAM,0,NDIM,NDRMAX,F,V,
        //      $           TOTMAS,0,WMAS,WMASS,
        //      $           EKCMTO,XCM,VCM,ACM,EKROT,OCM,ICMROT)

        //             CALL STOPCM(NATTOT,NDIM,F,V,XCM,VCM,OCM,ISCROT)
        //          ENDIF


        // removing the center of mass motion
        if(simParam.com && ((integrator.step % simParam.com) == 0)) {
            // cerr.printf("removeCOMMotion at step: %d\n", integrator.step);
            // cerr.flush();
            integrator.removeCOMMotion(conf, topology, simBox);
            // cerr.printf("removeCOMMotion\n");
            // cerr.flush();
        }

        {
            const number kineticEnergy = (conf.current->kineticEnergy+conf.old->kineticEnergy) * 0.5;
            const number potentialEnergy = nonbonded.energy + bonded.energy;

            // saving the energetic data
            energies.totalEnergy[integrator.step] = potentialEnergy + kineticEnergy;
            energies.kineticEnergy[integrator.step] = kineticEnergy;
            energies.potentialEnergy[integrator.step] = potentialEnergy;
            energies.nonbondedEnergy[integrator.step] = nonbonded.energy;
            energies.nonbondedVdwEnergy[integrator.step] = nonbonded.vdwEnergy;
            energies.nonbondedCoulEnergy[integrator.step] = nonbonded.coulEnergy;
            energies.nonbondedVdw14Energy[integrator.step] = nonbonded.vdw14Energy;
            energies.nonbondedCoul14Energy[integrator.step] = nonbonded.coul14Energy;
            energies.bondedEnergy[integrator.step] = bonded.energy;
            energies.bondedBondEnergy[integrator.step] = bonded.bondEnergy;
            energies.bondedBondAngleEnergy[integrator.step] = bonded.bondAngleEnergy;
            energies.bondedImproperDihedralEnergy[integrator.step] = 0.0;
            energies.bondedProperDihedralEnergy[integrator.step] = bonded.properDihedralEnergy;
            energies.temperature[integrator.step] = (conf.current->temperature+conf.old->temperature) * 0.5;
            energies.pressure[integrator.step] = barostat->pressure();
        }

        if(integrator.step && ((integrator.step % 100) == 0)) {
            const double tmpTime = getElapsed(totalProgramTimer);
            const double etaTime = tmpTime / integrator.step * simParam.nstlim - tmpTime;
            cerr.printf("\r%6.2f%% spent: %16.3f %s. ETA til finish: %16.3f %s. ETA total time: %16.3f %s. %8.3f ns/day", 100.0 * integrator.step/simParam.nstlim, tmpTime, timeUnit, etaTime, timeUnit, etaTime+tmpTime, timeUnit, (integrator.step * simParam.dt * nsdayconstant) / tmpTime);
            // cout.printf("Avg. # steps without pairlist update: %d\n", totalStepsWithoutPairListUpdate / numberPairListUpdate);
            // cout.printf("Max displacement: %f and halfRlistDiff: %f\n", pairlist.maxDisplacement, halfRlistDiff);
        }
        // cerr.printf("end of step %d\n", integrator.step);
        // cerr.flush();
        // omdfile.flush();

    }
    cerr.printf("\n");
    // **************************************************
    addToTime(mdLoopTimer, mdLoopTime);

    energies.computeAveragesAndStdDevs();

    omdfile.printf("############################################\n");
    omdfile.printf("#              A V E R A G E S             #\n");
    omdfile.printf("############################################\n");
    omdfile.printf("# Number of Steps                %d\n", simParam.nstlim);
    omdfile.printf("# Total Energy                  %+e\n", energies.avgTotalEnergy);
    omdfile.printf("#    Kinetic Energy                %+e\n", energies.avgKineticEnergy);
    omdfile.printf("#    Potential Energy              %+e\n", energies.avgPotentialEnergy);
    omdfile.printf("#       Nonbonded Energy              %+e\n", energies.avgNonbondedEnergy);
    omdfile.printf("#          Vdw Energy                    %+e\n", energies.avgNonbondedVdwEnergy);
    omdfile.printf("#          Coulomb Energy                %+e\n", energies.avgNonbondedCoulEnergy);
    omdfile.printf("#          Vdw_14 Energy                 %+e\n", energies.avgNonbondedVdw14Energy);
    omdfile.printf("#          Coulomb_14 Energy             %+e\n", energies.avgNonbondedCoul14Energy);
    omdfile.printf("#       Bonded Energy                    %+e\n", energies.avgBondedEnergy);
    omdfile.printf("#          Bond Energy                   %+e\n", energies.avgBondedBondEnergy);
    omdfile.printf("#          BondAngle Energy              %+e\n", energies.avgBondedBondAngleEnergy);
    omdfile.printf("#          Improper dihedral Energy      %+e\n", energies.avgBondedImproperDihedralEnergy);
    omdfile.printf("#          Proper dihedral Energy        %+e\n", energies.avgBondedProperDihedralEnergy);
    omdfile.printf("# Temperature                   %+e\n", energies.avgTemperature);
    omdfile.printf("# Pressure                      %+e\n", energies.avgPressure);
    omdfile.printf("\n\n");

    omdfile.printf("############################################\n");
    omdfile.printf("#          F L U C T U A T I O N S         #\n");
    omdfile.printf("############################################\n");
    omdfile.printf("# Number of Steps                %d\n", simParam.nstlim);
    omdfile.printf("# Total Energy                  %+e\n", energies.stdDevTotalEnergy);
    omdfile.printf("#    Kinetic Energy                %+e\n", energies.stdDevKineticEnergy);
    omdfile.printf("#    Potential Energy              %+e\n", energies.stdDevPotentialEnergy);
    omdfile.printf("#       Nonbonded Energy              %+e\n", energies.stdDevNonbondedEnergy);
    omdfile.printf("#          Vdw Energy                    %+e\n", energies.stdDevNonbondedVdwEnergy);
    omdfile.printf("#          Coulomb Energy                %+e\n", energies.stdDevNonbondedCoulEnergy);
    omdfile.printf("#          Vdw_14 Energy                 %+e\n", energies.stdDevNonbondedVdw14Energy);
    omdfile.printf("#          Coulomb_14 Energy             %+e\n", energies.stdDevNonbondedCoul14Energy);
    omdfile.printf("#       Bonded Energy                    %+e\n", energies.stdDevBondedEnergy);
    omdfile.printf("#          Bond Energy                   %+e\n", energies.stdDevBondedBondEnergy);
    omdfile.printf("#          BondAngle Energy              %+e\n", energies.stdDevBondedBondAngleEnergy);
    omdfile.printf("#          Improper dihedral Energy      %+e\n", energies.stdDevBondedImproperDihedralEnergy);
    omdfile.printf("#          Proper dihedral Energy        %+e\n", energies.stdDevBondedProperDihedralEnergy);
    omdfile.printf("# Temperature                   %+e\n", energies.stdDevTemperature);
    omdfile.printf("# Pressure                      %+e\n", energies.stdDevPressure);
    omdfile.printf("\n\n");

    addToTime(totalProgramTimer, totalProgramTime);

    omdfile.printf("-------------------------------------\n");
    omdfile.printf("                         # thread(s): %d\n", dd.nprocs);
    omdfile.printf("-------------------------------------\n");
    omdfile.printf("Avg. # steps without pairlist update: %d\n", totalStepsWithoutPairListUpdate / numberPairListUpdate);
    omdfile.printf("                  # pairlist updates: %d\n", numberPairListUpdate);
    omdfile.printf("-------------------------------------\n");
    omdfile.printf("               Print trajectory time: %16.3f %s %16.3f%%\n", printTRJTime,         timeUnit, 100.0 * printTRJTime/totalProgramTime);
    omdfile.printf("              PairList creation time: %16.3f %s %16.3f%%\n", pairListCreateTime,   timeUnit, 100.0 * pairListCreateTime/totalProgramTime);
    omdfile.printf("                PairList update time: %16.3f %s %16.3f%%\n", pairListUpdateTime,   timeUnit, 100.0 * pairListUpdateTime/totalProgramTime);

    omdfile.printf("           atomic neighbor list time: %16.3f %s %16.3f%%\n", updateAtomNeighListOfLocalCellsPairlistTime,   timeUnit, 100.0 * updateAtomNeighListOfLocalCellsPairlistTime/totalProgramTime);
    omdfile.printf("            clean neighbor list time: %16.3f %s %16.3f%%\n", resetLocalCellsAndTheAtomNeighListPairlistTime,   timeUnit, 100.0 * resetLocalCellsAndTheAtomNeighListPairlistTime/totalProgramTime);
    omdfile.printf("         place atoms into cells time: %16.3f %s %16.3f%%\n", placeAtomsIntoAllCellsPairlistTime,   timeUnit, 100.0 * placeAtomsIntoAllCellsPairlistTime/totalProgramTime);

    omdfile.printf("                 Kinetic Energy time: %16.3f %s %16.3f%%\n", kineticEnergyTime,    timeUnit, 100.0 * kineticEnergyTime/totalProgramTime);
    omdfile.printf("              Nonbonded compute time: %16.3f %s %16.3f%%\n", nonbondedComputeTime, timeUnit, 100.0 * nonbondedComputeTime/totalProgramTime);
    omdfile.printf("                Nonbonded first time: %16.3f %s %16.3f%%\n", nonbondedFirstTime,   timeUnit, 100.0 * nonbondedFirstTime/totalProgramTime);
    omdfile.printf("               Nonbonded second time: %16.3f %s %16.3f%%\n", nonbondedSecondTime,  timeUnit, 100.0 * nonbondedSecondTime/totalProgramTime);
    omdfile.printf("            Nonbonded reduction time: %16.3f %s %16.3f%%\n", nonbondedReductionTime,  timeUnit, 100.0 * nonbondedReductionTime/totalProgramTime);
    omdfile.printf("                           Bond time: %16.3f %s %16.3f%%\n", bondTime,   timeUnit, 100.0 * bondTime/totalProgramTime);
    omdfile.printf("------------------------------------- %35.3f%%\n", 100.0 * (printTRJTime + pairListCreateTime + pairListUpdateTime + kineticEnergyTime + nonbondedComputeTime + bondTime)/totalProgramTime);
    omdfile.printf("                      Integrate time: %16.3f %s %16.3f%%\n", integrationTime,      timeUnit, 100.0 * integrationTime/totalProgramTime);
    omdfile.printf("                             MD time: %16.3f %s %16.3f%%\n", mdLoopTime,           timeUnit, 100.0 * mdLoopTime/totalProgramTime);
    omdfile.printf("-------------------------------------\n");
    omdfile.printf("                          Total time: %16.3f %s %16.3f%%\n", totalProgramTime, timeUnit, 100.0);
    omdfile.printf("                              ns/day: %16.3f\n", (simParam.nstlim * simParam.dt * nsdayconstant) / (totalProgramTime));
    omdfile.printf("-------------------------------------\n");

    cout.printf("-------------------------------------\n");
    cout.printf("                         # thread(s): %d\n", dd.nprocs);
    cout.printf("-------------------------------------\n");
    cout.printf("Avg. # steps without pairlist update: %d\n", totalStepsWithoutPairListUpdate / numberPairListUpdate);
    cout.printf("                  # pairlist updates: %d\n", numberPairListUpdate);
    cout.printf("-------------------------------------\n");
    cout.printf("               Print trajectory time: %16.3f %s %16.3f%%\n", printTRJTime,         timeUnit, 100.0 * printTRJTime/totalProgramTime);
    cout.printf("              PairList creation time: %16.3f %s %16.3f%%\n", pairListCreateTime,   timeUnit, 100.0 * pairListCreateTime/totalProgramTime);
    cout.printf("                PairList update time: %16.3f %s %16.3f%%\n", pairListUpdateTime,   timeUnit, 100.0 * pairListUpdateTime/totalProgramTime);

    cout.printf("           atomic neighbor list time: %16.3f %s %16.3f%%\n", updateAtomNeighListOfLocalCellsPairlistTime,   timeUnit, 100.0 * updateAtomNeighListOfLocalCellsPairlistTime/totalProgramTime);
    cout.printf("            clean neighbor list time: %16.3f %s %16.3f%%\n", resetLocalCellsAndTheAtomNeighListPairlistTime,   timeUnit, 100.0 * resetLocalCellsAndTheAtomNeighListPairlistTime/totalProgramTime);
    cout.printf("         place atoms into cells time: %16.3f %s %16.3f%%\n", placeAtomsIntoAllCellsPairlistTime,   timeUnit, 100.0 * placeAtomsIntoAllCellsPairlistTime/totalProgramTime);

    cout.printf("                 Kinetic Energy time: %16.3f %s %16.3f%%\n", kineticEnergyTime,    timeUnit, 100.0 * kineticEnergyTime/totalProgramTime);
    cout.printf("              Nonbonded compute time: %16.3f %s %16.3f%%\n", nonbondedComputeTime, timeUnit, 100.0 * nonbondedComputeTime/totalProgramTime);
    cout.printf("                Nonbonded first time: %16.3f %s %16.3f%%\n", nonbondedFirstTime,   timeUnit, 100.0 * nonbondedFirstTime/totalProgramTime);
    cout.printf("               Nonbonded second time: %16.3f %s %16.3f%%\n", nonbondedSecondTime,  timeUnit, 100.0 * nonbondedSecondTime/totalProgramTime);
    cout.printf("            Nonbonded reduction time: %16.3f %s %16.3f%%\n", nonbondedReductionTime,  timeUnit, 100.0 * nonbondedReductionTime/totalProgramTime);
    cout.printf("                           Bond time: %16.3f %s %16.3f%%\n", bondTime,   timeUnit, 100.0 * bondTime/totalProgramTime);
    cout.printf("------------------------------------- %35.3f%%\n", 100.0 * (printTRJTime + pairListCreateTime + pairListUpdateTime + kineticEnergyTime + nonbondedComputeTime + bondTime)/totalProgramTime);
    cout.printf("                      Integrate time: %16.3f %s %16.3f%%\n", integrationTime,      timeUnit, 100.0 * integrationTime/totalProgramTime);
    cout.printf("                             MD time: %16.3f %s %16.3f%%\n", mdLoopTime,           timeUnit, 100.0 * mdLoopTime/totalProgramTime);
    cout.printf("-------------------------------------\n");
    cout.printf("                          Total time: %16.3f %s %16.3f%%\n", totalProgramTime, timeUnit, 100.0);
    cout.printf("                              ns/day: %16.3f\n", (simParam.nstlim * simParam.dt * nsdayconstant) / (totalProgramTime));
    cout.printf("-------------------------------------\n");

    outputTRAJFile->close();

    omdfile.printf("Normal Termination\n");
    // omdfile << "Normal Termination\n";
    std::cout << "Normal Termination\n";

    dd.finalize();

    return 0;
}

/* VELOCIT VERLET LOOP

    // **************************************************
    // main MD loop
    startTimer(mdLoopTimer);
    for(integrator.step = 1; integrator.step <= simParam.nstlim; integrator.step++) {


        // write output, if requested
        if (simParam.ntpr && ((integrator.step % simParam.ntpr) == 0)) {
            omdfile.printStep(simParam, nonbonded, simBox, integrator);

        // pairlist.print(omdfile, integrator.step);
        }

        if (simParam.ntwx && ((integrator.step % simParam.ntwx) == 0)) {
            ss << "time step: " << integrator.step;
            conf.title = ss.str();
            ss.str(std::string());

            // printGRO(trjfile, conf, simBox);
            outputTRAJFile->writeFile(conf, simBox);
        }

        // propagate system and recompute energies
        integrator.integrate(conf, topology, pairlist, simParam, nonbonded, simBox);
        // compute the kinetic energy
        integrator.computeKineticEnergyAndTemperature(conf, topology, simParam);

        // thermostating if requested
        if(thermostat != NULL)
            thermostat->apply(integrator.kineticEnergy, conf);


        // update cell list
        if(pairlist.doMaxDisplacement) {

            if(pairlist.maxDisplacement > halfRlistDiff) {

                // Update statistics on the pairlist update
                totalStepsWithoutPairListUpdate += stepsWithoutPairListUpdate;
                stepsWithoutPairListUpdate=1;
                numberPairListUpdate++;

                // omdfile.printf("\npairlist.maxDisplacement: %f, halfRlistDiff: %f\n", pairlist.maxDisplacement, halfRlistDiff);
                // omdfile.printf("Avg. # steps without pairlist update: %d\n", totalStepsWithoutPairListUpdate / numberPairListUpdate);

                // cerr.printf("\npairlist.maxDisplacement: %f, halfRlistDiff: %f\n", pairlist.maxDisplacement, halfRlistDiff);
                // cerr.printf("Avg. # steps without pairlist update: %d\n", totalStepsWithoutPairListUpdate / numberPairListUpdate);

                pairlist.update(conf, simBox, topology);
            }
            else
                stepsWithoutPairListUpdate++;

        } else if (integrator.step % simParam.nstlist == 0) {

            // Update statistics on the pairlist update
            totalStepsWithoutPairListUpdate += stepsWithoutPairListUpdate;
            stepsWithoutPairListUpdate=1;
            numberPairListUpdate++;
            // cerr.printf("pairlist.maxDisplacement: %f, rlistDiff: %f\n", pairlist.maxDisplacement, 0.5 * rlistDiff);
            // cerr.printf("Avg. # steps without pairlist update: %d\n", totalStepsWithoutPairListUpdate / numberPairListUpdate);
            pairlist.update(conf, simBox, topology);

        }
        else
        {
            stepsWithoutPairListUpdate++;
        }

        // removing the center of mass motion
        if(simParam.com && ((integrator.step % simParam.com) == 0))
            integrator.removeCOMMotion(conf, topology);

        if((integrator.step % 100) == 0) {
            const double tmpTime = getElapsed(totalProgramTimer);
            const double etaTime = tmpTime / integrator.step * simParam.nstlim - tmpTime;
            cerr.printf("\r%6.2f%% spent: %16.3f %s. ETA til finish: %16.3f %s. ETA total time: %16.3f %s. %8.3f ns/day", 100.0 * integrator.step/simParam.nstlim, tmpTime, timeUnit, etaTime, timeUnit, etaTime+tmpTime, timeUnit, (integrator.step * simParam.dt * nsdayconstant) / tmpTime);
            // cout.printf("Avg. # steps without pairlist update: %d\n", totalStepsWithoutPairListUpdate / numberPairListUpdate);
            // cout.printf("Max displacement: %f and halfRlistDiff: %f\n", pairlist.maxDisplacement, halfRlistDiff);
        }
    }
    */
